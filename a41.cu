#include<iostream>  
#include<hip/hip_runtime.h>

__global__ void add(int *a, int *b, int *c,int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}

int main(){

    int n = 10;
    int *a_cpu = new int[n];
    int *b_cpu = new int[n];
    int *c_cpu = new int[n];

    // fill arrays with random numbers
    for (int i = 0; i < n; i++) {
        a_cpu[i] = rand() % 100;
        b_cpu[i] = rand() % 100;
    }

    // declare GPU memory pointers
    int *a_gpu, *b_gpu, *c_gpu;

    // allocate GPU memory
    hipMalloc(&a_gpu, n * sizeof(int));
    hipMalloc(&b_gpu, n * sizeof(int));
    hipMalloc(&c_gpu, n * sizeof(int));

    // transfer the array to the GPU
    hipMemcpy(a_gpu, a_cpu, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(b_gpu, b_cpu, n * sizeof(int), hipMemcpyHostToDevice);


    // define grid and block size
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;

    // launch the kernel
    add<<<gridSize, blockSize>>>(a_gpu, b_gpu, c_gpu, n);

    hipMemcpy(c_cpu, c_gpu, n * sizeof(int), hipMemcpyDeviceToHost);

    // print out the results
    for (int i = 0; i < n; i++) {
        std::cout << a_cpu[i] << " + " << b_cpu[i] << " = " << c_cpu[i] << std::endl;
    }

    // free GPU memory allocation
    hipFree(a_gpu);
    hipFree(b_gpu);
    hipFree(c_gpu);

    // free CPU memory allocation
    delete[] a_cpu;
    delete[] b_cpu;
    delete[] c_cpu;

    return 0;

}


































// The provided code is a CUDA program that performs element-wise addition of two arrays `a` and `b` on the GPU using parallel execution.

// Here's an explanation of the code:

// 1. The code includes necessary header files: `iostream` and `cuda.h`.

// 2. The `add` kernel function is defined. It takes three integer array pointers `a`, `b`, and `c`, along with an integer `n` representing the array size. The kernel is executed by each thread, where the `index` is calculated based on the thread and block indices. The if condition ensures that threads outside the array bounds do not perform any computation. Each thread adds the corresponding elements from `a` and `b` and stores the result in `c`.

// 3. In the `main` function:
//    - The array size `n` is set to 10.
//    - Three integer arrays `a_cpu`, `b_cpu`, and `c_cpu` are created on the host (CPU) to store the input arrays and the result.
//    - The arrays `a_cpu` and `b_cpu` are filled with random numbers.
//    - Pointers for GPU memory allocation, `a_gpu`, `b_gpu`, and `c_gpu`, are declared.
//    - Memory is allocated on the GPU using `cudaMalloc` for `a_gpu`, `b_gpu`, and `c_gpu`.
//    - The input arrays `a_cpu` and `b_cpu` are transferred from the CPU to the GPU using `cudaMemcpy`.
//    - The block size and grid size are set to 256 and `(n + blockSize - 1) / blockSize` respectively, to configure the number of threads.
//    - The `add` kernel is launched using `<<<gridSize, blockSize>>>`, passing the GPU array pointers and the size `n`.
//    - The result array `c_gpu` is copied back from the GPU to the CPU using `cudaMemcpy`.
//    - Finally, the result is printed by iterating over the arrays `a_cpu`, `b_cpu`, and `c_cpu`.

// 4. Memory allocated on the GPU is freed using `cudaFree`.

// 5. Memory allocated on the CPU is released using `delete[]`.

// The code demonstrates how to perform array addition in parallel using CUDA. It utilizes GPU parallelism to perform the addition operation on multiple elements simultaneously, potentially achieving better performance compared to a sequential CPU implementation.